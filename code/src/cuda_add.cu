#include "hip/hip_runtime.h"
extern "C" __constant__ int my_constant 314;
extern "C" __global__ void sum (const float* x, const float* y, float* out, int count){
        for (int i(((threadIdx.x)+(((blockIdx.x)*(blockDim.x)))));i<count;(i)++) {
                        out[i]=((x[i])*(y[i]));
}
}